#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <float.h>
#include "MnistPreProcess.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <string.h>

#define FEAT_KEY 0
#define CUT_KEY 1
#define LEFT_KEY 2
#define RIGHT_KEY 3
#define PRED_KEY 4
#define DEPTH_KEY 5

#define NUM_FIELDS 6

#define index(i, j, N)  ((i)*(N)) + (j)
#define index(i, j, N)  ((i)*(N)) + (j)
#define ixt(i, j, t, N, T) ((t)*(N)*(T)) + ((i)*(N)) + (j)
#define MIN(a,b) (((a)<(b))?(a):(b))
#define MAX(a,b) (((a)>(b))?(a):(b))
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
 
int countNumRows(char *filename)
{
	FILE *fp;
	int count = 0;  // Line counter (result)
	//char filename[MAX_FILE_NAME];
	char c;  // To store a character read from file
 
	// Get file name from user. The file should be
	// either in current folder or complete path should be provided
	//printf("Enter file name: ");
	//scanf("%s", filename);
 
	// Open the file
	fp = fopen(filename, "r");
 
	// Check if file exists
	if (fp == NULL)
	{
		printf("Could not open file %s", filename);
		return -1;
	}
 
	// Extract characters from file and store in character c
	for (c = getc(fp); c != EOF; c = getc(fp))
		if (c == '\n') // Increment count if this character is newline
			count = count + 1;
 
	// Close the file
	fclose(fp);
	//printf("The file %s has %d lines\n ", filename, count);
 
	return count;
}

const char* getfield(char* line, int num){
	const char* tok;
	for (tok = strtok(line, ",");
			tok && *tok;
			tok = strtok(NULL, ",\n"))
	{
		if (!--num)
			return tok;
	}
	return NULL;
}
/*
Labels for IRIS:
Iris-setosa - 0
Iris-versicolor - 1
Iris-virginica - 2
*/
void read_csv_iris(float *data, float *label, int row_count, char *filename){
	//data = (float *)malloc(row_count*4*sizeof(float));
	//label = (int *)malloc(row_count*sizeof(int));
	FILE *fp = fopen(filename,"r");
	char line[1024];
	int idx = 0;
	for(int iter = 0;iter<row_count;iter++)
	{
		fgets(line,1024,fp);
		const char *temp_field;
		for(int i=0;i<5;i++)
		{
			float temp_num;
			char *tmp = strdup(line);
			temp_field = getfield(tmp,i+1);
			if(i==4)
			{
				if(strcmp(temp_field,"Iris-setosa")==0)
				{
					label[idx] = 0;
					continue;
				}
				if(strcmp(temp_field,"Iris-versicolor")==0)
				{
					label[idx] = 1;
					continue;
				}
				if(strcmp(temp_field,"Iris-virginica")==0)
				{
					label[idx] = 2;
					continue;
				}
			}
			temp_num = atof(temp_field);
			data[idx*4 + i] = temp_num;
		}
		idx++;
		
	}
} 

/**
 * assume csv file fields separated by commas
 * get number of features, for allocating storage
 * @param filename: name of csv file to count features in
 * return: number of features in file
 */
int get_num_features(char *filename){
  FILE *fp = fopen(filename, "r");
  char *line = NULL;
  size_t len = 0;
  int result = 0;
  char delim[] = ","; // assume file separated by commas
  getline(&line, &len, fp);
  char *ptr = strtok(line, delim);
  while (ptr != NULL){
    ptr = strtok(NULL, delim);
    result++;
  }
  free(line);
  return result;
}

void read_csv(float *data, float *label, int row_count, char *filename){
  FILE *fp = fopen(filename,"r");
  assert(fp != NULL);
  size_t len = 0;
  char *line = NULL;
  int charsRead = 0;
  charsRead = getline (&line, &len, fp);
  int i = 0;
  char delim[] = ","; // assume file separated by commas
  float curFloat; // holds current floating point number read from file
  while (charsRead > 0){
    // according to man page, should free buffer read
    free(line);
    line = NULL;
    char *ptr = strtok(line, delim);
    while (ptr != NULL){
      ptr = strtok(NULL, delim);
      sscanf(ptr, "%f", &curFloat);
    }
    charsRead = getline (&line, &len, fp);
    i++;
  }
  printf("read %d lines\n", i);
  printf("%s", line);
  assert(false);
}

void readData(float* dataset,float*labels,const char* dataPath,const char*labelPath)
{
	FILE* dataFile=fopen(dataPath,"rb");
	FILE* labelFile=fopen(labelPath,"rb");
	int mbs=0,number=0,col=0,row=0;
	fread(&mbs,4,1,dataFile);
	fread(&number,4,1,dataFile);
	fread(&row,4,1,dataFile);
	fread(&col,4,1,dataFile);
	revertInt(&mbs);
	revertInt(&number);
	revertInt(&row);
	revertInt(&col);
	fread(&mbs,4,1,labelFile);
	fread(&number,4,1,labelFile);
	revertInt(&mbs);
	revertInt(&number);
	unsigned char temp;
	for(int i=0;i<number;++i)
	{
		for(int j=0;j<row*col;++j)
		{
			fread(&temp,1,1,dataFile);
			//dataset[i][j]=static_cast<float>(temp);
			dataset[(i*row*col) + j] = (float)temp;
		}
		fread(&temp,1,1,labelFile);
		//printf("%s\n",*temp );
		//labels[i]=static_cast<float>(temp);
		labels[i] = (float)temp;
		//printf("%f\n", labels[i]);
	}
	fclose(dataFile);
	fclose(labelFile);
}


/* === Utils === */
int next_pow_2(int x){
	int y = 1;
	while(y < x)
		y*=2;
	return y;
}
void copy_transpose(float* to, float* from, int h, int w){
	for(int i=0; i<h; i++){
		for(int j=0; j<w; j++){
			to[index(j, i, h)] = from[index(i, j, w)];
		}
	}
}
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
	// From https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-
	//   to-check-for-errors-using-the-cuda-runtime-api
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
void debug(){
	hipError_t code;
	code = hipPeekAtLastError();
	if(code != hipSuccess){
		printf("GPUassert: Failed at Init: %s\n", hipGetErrorString(code));
		exit(code);
	}
	code = hipDeviceSynchronize();
	if(code != hipSuccess){
		printf("GPUassert: Failed at Execution: %s\n", hipGetErrorString(code));
		exit(code);
	}
}

/* === Random Init === */
__global__ void init_random(unsigned int seed, hiprandState_t* states) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(seed, tid, 0, &states[tid]);
}
__device__ int draw_approx_binomial(int n, float p, hiprandState_t* state) {
	int x = (int) round(hiprand_normal(state) * n*p*(1-p) + n*p);
	return max(0, min(x, n));
}
__device__ float draw_uniform(float minimum, float maximum, hiprandState_t* state){
	return minimum + hiprand_uniform(state) * (maximum - minimum);
}
__device__ int draw_uniform_int(int minimum, int maximum, hiprandState_t* state){
	return floor(draw_uniform(minimum, maximum, state));
}

/* === Expanding tree memory === */
void expand(float** d_trees_ptr, int num_trees, int tree_arr_length, int new_tree_arr_length){
	float *new_d_trees, *d_trees;
	d_trees = *d_trees_ptr;
	assert(new_tree_arr_length >= tree_arr_length);

	hipMalloc((void **) &new_d_trees, num_trees * NUM_FIELDS * new_tree_arr_length * sizeof(float));
	for(int i=0; i<num_trees; i++){
		hipMemcpy(
			new_d_trees + i * (NUM_FIELDS * new_tree_arr_length), 
			d_trees + i * (NUM_FIELDS * tree_arr_length),  
			(NUM_FIELDS * tree_arr_length) * sizeof(float), hipMemcpyDeviceToDevice);
	}
	hipFree(d_trees);
	*d_trees_ptr = new_d_trees;
}
__global__ void get_max_tree_length(int* d_tree_lengths, int num_trees, int* d_max_tree_length){
	extern __shared__ int tree_length_buffer[];
	if(threadIdx.x < num_trees){
		tree_length_buffer[threadIdx.x] = d_tree_lengths[threadIdx.x];
	}else{
		tree_length_buffer[threadIdx.x] = -1;
	}
	
	for(int stride=blockDim.x/2; stride > 0; stride >>=1){
		__syncthreads();
		if(threadIdx.x < stride){
			if(tree_length_buffer[threadIdx.x + stride] > tree_length_buffer[threadIdx.x]){
				tree_length_buffer[threadIdx.x] = tree_length_buffer[threadIdx.x + stride];
			}
		}
	}
	if(threadIdx.x == 0){
	   d_max_tree_length[0] = tree_length_buffer[0];
	}
}
void maybe_expand(float** d_trees_ptr, int num_trees, int* tree_arr_length, int* d_tree_lengths,
	                int* max_tree_length, int* d_max_tree_length){
	// I wonder if it's faster just to compute max on CPU.
	int new_tree_arr_length;

	get_max_tree_length<<<1, next_pow_2(num_trees), next_pow_2(num_trees) * sizeof(int)>>>(
		d_tree_lengths, num_trees, d_max_tree_length
	);
	hipMemcpy(max_tree_length, d_max_tree_length, sizeof(int), hipMemcpyDeviceToHost);

	// Buffer of 2 => up to 2 additions at a time
	if(*max_tree_length <= *tree_arr_length-4){
		return;
	}else{
		new_tree_arr_length = (*tree_arr_length) * 2;
        while(*max_tree_length > new_tree_arr_length-3){
            new_tree_arr_length *= 2;
        }

        printf("Expanding to %d\n", new_tree_arr_length);
        expand(d_trees_ptr, num_trees, *tree_arr_length, new_tree_arr_length);
        *tree_arr_length = new_tree_arr_length;
	}
}

/* === Tree Initialization === */
__global__ void kernel_initialize_trees(float *d_trees, int* d_tree_lengths, int tree_arr_length){
	d_trees[ixt(0, LEFT_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] = 0;
	d_trees[ixt(0, RIGHT_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] = 0;
	d_trees[ixt(0, DEPTH_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] = 0;
	d_trees[ixt(0, PRED_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] = -1;
	d_tree_lengths[threadIdx.x] = 1;
}
void initialize_trees(float* d_trees, int num_trees, int tree_arr_length, int* d_tree_lengths){
	kernel_initialize_trees<<<1, num_trees>>>(d_trees, d_tree_lengths, tree_arr_length);
}
__global__ void kernel_initialize_batch_pos(int *d_batch_pos, int x_length, int num_trees){
	int i;
	for(i=threadIdx.x; i<x_length; i+=blockDim.x){
		d_batch_pos[index(blockIdx.x, i, x_length)] = 0;
	}
}
void initialize_batch_pos(int *d_batch_pos, int x_length, int num_trees, hipDeviceProp_t dev_prop){
	kernel_initialize_batch_pos<<<num_trees, dev_prop.maxThreadsPerBlock>>>(
		d_batch_pos, x_length, num_trees
	);
}

/* === Tree Growth checks === */
__global__ void kernel_refresh_tree_is_done(int* d_tree_lengths, int* d_tree_is_done, int tree_pos){
	// threadIdx.x = tree_id
	int is_done;
	if(tree_pos < d_tree_lengths[threadIdx.x]){
		is_done = 0;
	}else{
		is_done = 1;
	}
	d_tree_is_done[threadIdx.x] = is_done;
}
void refresh_tree_is_done(int* d_tree_lengths, int* d_tree_is_done, int tree_pos, int num_trees){
	kernel_refresh_tree_is_done<<<1, num_trees>>>(
		d_tree_lengths, d_tree_is_done, tree_pos
	);
}
int check_forest_done(int* d_tree_is_done, int *tree_is_done, int num_trees){
	hipMemcpy(tree_is_done, d_tree_is_done, num_trees * sizeof(int), hipMemcpyDeviceToHost);
	int trees_left;
	trees_left = 0;
	for(int i=0; i<num_trees; i++){
		if(!tree_is_done[i]){
			trees_left++;
		}
	}
	printf("%d trees left to grow\n", trees_left);
	if(trees_left == 0){
		return 1;
	}else{
		return 0;
	}
}

/* === Tree Traversal === */
__global__ void kernel_traverse_trees(
			float *d_trees, float* d_x, 
			int x_length, int num_trees, int tree_arr_length, 
			int* d_batch_pos, int FEATURE
		){
	// Should optimize this. It's just a bunch of global reads.
	// Also possibly to rewrite this and batch_traverse to support a "next-step" method instead of a full 
	//   traversal while growing
	int pos, new_pos, left_right_key, x_i, tree_id, tx;
	tx = threadIdx.x + blockIdx.x * blockDim.x;
	if(tx >= x_length * num_trees) return;

	// Actually get x_i, tree_id
	tree_id = tx % num_trees;
	x_i = tx / num_trees;
	pos = 0;
    while(1){
        if(d_x[index(x_i, (int) d_trees[
        		ixt(pos, FEAT_KEY, tree_id, NUM_FIELDS, tree_arr_length)], FEATURE)] < 
    				d_trees[ixt(pos, CUT_KEY, tree_id, NUM_FIELDS, tree_arr_length)]){
            left_right_key = LEFT_KEY;
        }else{
            left_right_key = RIGHT_KEY;
        }
        new_pos = (int) d_trees[ixt(pos, left_right_key, tree_id, NUM_FIELDS, tree_arr_length)];
        if(new_pos == pos){
            // Leaf nodes are set up to be idempotent
            break;
        }
        pos = new_pos;
    }
    d_batch_pos[index(tree_id, x_i, x_length)] = pos;
}
void batch_traverse_trees(
			float *d_trees, float *d_x, 
			int x_length, int num_trees, int tree_arr_length, 
			int *d_batch_pos, hipDeviceProp_t dev_prop, int FEATURE){
	int block_size, num_blocks;
 	block_size = dev_prop.maxThreadsPerBlock;
 	num_blocks = ceil(num_trees * x_length/((float) block_size));
	kernel_traverse_trees<<<num_blocks, block_size>>>(
		d_trees, d_x, x_length, num_trees, tree_arr_length, d_batch_pos, FEATURE
	);
}
__global__ void kernel_advance_trees(
			float *d_trees, float* d_x, int x_length, int tree_arr_length, 
			int num_trees, int* d_batch_pos, int TRAIN_NUM, int FEATURE
		){
	int pos, left_right_key, x_i;
	// threadIdx.x = x_i, blockIdx.x = tree_id
	for(x_i=threadIdx.x; x_i < x_length; x_i+=blockDim.x){
		pos = d_batch_pos[index(blockIdx.x, x_i, TRAIN_NUM)];
	    if(d_x[index(x_i, (int) d_trees[
	    	ixt(pos, FEAT_KEY, blockIdx.x, NUM_FIELDS, tree_arr_length)], FEATURE)] < 
	    		d_trees[ixt(pos, CUT_KEY, blockIdx.x, NUM_FIELDS, tree_arr_length)]){
	        left_right_key = LEFT_KEY;
	    }else{
	        left_right_key = RIGHT_KEY;
	    }
	    d_batch_pos[index(blockIdx.x, x_i, TRAIN_NUM)] = 
	    	(int) d_trees[ixt(pos, left_right_key, blockIdx.x, NUM_FIELDS, tree_arr_length)];
	}
}
void batch_advance_trees(
			float *d_tree, float *d_x, int x_length, 
			int tree_arr_length, int num_trees, int *d_batch_pos, 
			hipDeviceProp_t dev_prop, int TRAIN_NUM, int FEATURE
		){
	kernel_advance_trees<<<num_trees, dev_prop.maxThreadsPerBlock>>>(
		d_tree, d_x, x_length, tree_arr_length, num_trees, d_batch_pos, TRAIN_NUM, FEATURE
	);
}

/* === Node termination === */
__global__ void kernel_check_node_termination(
			float* d_trees, int tree_arr_length,
			float* d_y, int* d_batch_pos, int tree_pos, 
			int* d_is_branch_node, int* d_tree_is_done, int TRAIN_NUM
		){
	// threadIdx.x = tree_id
	int i, base_y, new_y, is_branch_node;

	// If tree is done, it's never a branch node
	if(d_tree_is_done[threadIdx.x]==1){
		d_is_branch_node[threadIdx.x] = 0;
		return;
	}

	// Check for non-unique Y
	base_y = -1;
	is_branch_node = 0;
	for(i=0; i<TRAIN_NUM; i++){
		if(d_batch_pos[index(threadIdx.x, i, TRAIN_NUM)] == tree_pos){
			new_y = d_y[i];
			if(base_y == -1){
				base_y = new_y;
			}else if(base_y != new_y){
				is_branch_node = 1;
				break;
			}
		}
	}
	d_is_branch_node[threadIdx.x] = is_branch_node;

	if(base_y==-1){
		printf("ERROR EMPTY TREE %d\n", threadIdx.x);
		assert(false);
	}

	if(!is_branch_node){
		d_trees[ixt(tree_pos, PRED_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] = base_y;
	}
}
void check_node_termination(
			float* d_trees, int tree_arr_length,
			float* d_y, int* d_batch_pos, int tree_pos, 
			int* d_is_branch_node, int* d_tree_is_done,
			int num_trees, int TRAIN_NUM
		){
	kernel_check_node_termination<<<1, num_trees>>>(
		d_trees, tree_arr_length, 
		d_y, d_batch_pos, tree_pos,
		d_is_branch_node, d_tree_is_done, TRAIN_NUM
	);
	debug();
}

/* === Valid features === */
__global__ void kernel_collect_min_max(float* d_x_T, int* d_batch_pos, int desired_pos, int num_trees, 
									   int x_length, float* d_min_max_buffer, int* d_is_branch_node,int TRAIN_NUM,int FEATURE){
	extern __shared__ float shared_min_max[]; // threadIdx.x * 2
	// Ripe for optimization.
	// threadIdx.x = x_i++, blockIdx.x = tree_id, feat = blockIdx.y
	int x_i;
	float minimum, maximum, val;

	if(!d_is_branch_node[blockIdx.x]){
		return;
	}

	minimum = FLT_MAX;
	maximum = -FLT_MAX;
	for(x_i=threadIdx.x; x_i < x_length; x_i+=blockDim.x){
		if(d_batch_pos[index(blockIdx.x, x_i, x_length)] == desired_pos){
			val = d_x_T[index(blockIdx.y, x_i, TRAIN_NUM)];
			if(val < minimum){
				minimum = val;
			}
			if(val > maximum){
				maximum = val;
			}
		}
	}
	shared_min_max[index(threadIdx.x, 0, 2)] = minimum;
	shared_min_max[index(threadIdx.x, 1, 2)] = maximum;

	for(int stride=blockDim.x/2; stride > 0; stride >>=1){
		__syncthreads();
		if(threadIdx.x < stride){
			if(shared_min_max[index(threadIdx.x + stride, 0, 2)] < 
					shared_min_max[index(threadIdx.x, 0, 2)]){
				shared_min_max[index(threadIdx.x, 0, 2)] = 
					shared_min_max[index(threadIdx.x + stride, 0, 2)];
			}
			if(shared_min_max[index(threadIdx.x + stride, 1, 2)] > 
					shared_min_max[index(threadIdx.x, 1, 2)]){
				shared_min_max[index(threadIdx.x, 1, 2)] = 
					shared_min_max[index(threadIdx.x + stride, 1, 2)];
			}
		}
	}
	if(threadIdx.x==0){
		d_min_max_buffer[ixt(blockIdx.y, 0, blockIdx.x, 2, FEATURE)] = shared_min_max[index(0, 0, 2)];
		d_min_max_buffer[ixt(blockIdx.y, 1, blockIdx.x, 2, FEATURE)] = shared_min_max[index(0, 1, 2)];
	}
}
void collect_min_max(float* d_x_T, int* d_batch_pos, int desired_pos, int num_trees, int x_length,
					 float* d_min_max_buffer, int* d_is_branch_node, hipDeviceProp_t dev_prop,int TRAIN_NUM,int FEATURE){
	// Ripe for optimization.
	dim3 grid(num_trees, FEATURE);
	kernel_collect_min_max<<<grid, 64, 64 * sizeof(int) * 2>>>(
		d_x_T, d_batch_pos, desired_pos, num_trees, x_length, d_min_max_buffer, d_is_branch_node, TRAIN_NUM, FEATURE
	);	
}
__global__ void kernel_collect_num_valid_feat(
			int* d_num_valid_feat, int* d_random_feats_idx,
			float* d_min_max_buffer, int num_trees, int* d_is_branch_node, int FEATURE
		){
	int feat_i, tree_id, num_valid_feat;
	tree_id = threadIdx.x + blockIdx.x * blockDim.x;
	if(tree_id >= num_trees){
		return;
	}

	num_valid_feat = 0;
	for(feat_i=0; feat_i<FEATURE; feat_i++){
		if(d_min_max_buffer[ixt(feat_i, 0, tree_id, 2, FEATURE)] != 
			d_min_max_buffer[ixt(feat_i, 1, tree_id, 2, FEATURE)]
			){
			d_random_feats_idx[index(tree_id, num_valid_feat, FEATURE)] = feat_i;
			num_valid_feat++;
		}
	}
	d_num_valid_feat[tree_id] = num_valid_feat;
}
void collect_num_valid_feat(
			int* d_num_valid_feat, 
			int* d_random_feats_idx, 
			float* d_min_max_buffer, int num_trees, int* d_is_branch_node, 
			hipDeviceProp_t dev_prop, int FEATURE
		){
	// Ripe for optimization
	int grid_size = (int) ceil(1.0 * num_trees / 64);
	int block_size = 64;
	kernel_collect_num_valid_feat<<<grid_size, block_size>>>(
		d_num_valid_feat, d_random_feats_idx,
		d_min_max_buffer, num_trees, d_is_branch_node, FEATURE
	);
}

/* === Populate Random Features === */
__global__ void kernel_populate_feat_cut(
			int* d_random_feats, float* d_random_cuts,
			int* d_random_feats_idx, int* d_num_valid_feat, 
			float* d_min_max_buffer, int feat_per_node,
			int num_trees, int* d_is_branch_node, hiprandState_t* curand_states, int FEATURE
		){
	int i, num_valid_feat, feat_i, x, tree_id;
	tree_id = threadIdx.x + blockIdx.x * blockDim.x;
	float minimum, maximum;
	if(!d_is_branch_node[tree_id]){
		return;
	}
	num_valid_feat = d_num_valid_feat[tree_id];
	for(i=0; i<feat_per_node; i++){
		x = draw_uniform_int(0, num_valid_feat, curand_states+tree_id);
		feat_i = d_random_feats_idx[index(tree_id, x, FEATURE)];
		minimum = d_min_max_buffer[ixt(feat_i, 0, tree_id, 2, FEATURE)];
		maximum = d_min_max_buffer[ixt(feat_i, 1, tree_id, 2, FEATURE)];
		d_random_feats[index(tree_id, i, feat_per_node)] = feat_i;
		d_random_cuts[index(tree_id, i, feat_per_node)] = 
			draw_uniform(minimum, maximum, curand_states+tree_id);
	}
}
void populate_feat_cut(int* d_random_feats, float* d_random_cuts,
					   int* d_random_feats_idx, int* d_num_valid_feat,
	 				   float* d_min_max_buffer, int feat_per_node,
	 				   int num_trees, int* d_is_branch_node, hiprandState_t* curand_states,int FEATURE){
	int grid_size = (int) ceil(1.0 * num_trees / 64);
	int block_size = 64;
	kernel_populate_feat_cut<<<grid_size, block_size>>>(
		d_random_feats, d_random_cuts, 
		d_random_feats_idx, d_num_valid_feat, 
		d_min_max_buffer, feat_per_node, num_trees, 
		d_is_branch_node, curand_states, FEATURE
	);
}

/* === Count Classes === */
__global__ void kernel_populate_class_counts(
		float* d_x, float* d_y, int* d_class_counts_a, int* d_class_counts_b, 
		int* d_random_feats, float* d_random_cuts,
		int* d_batch_pos, int tree_pos,
		int num_trees, int feat_per_node, int* d_is_branch_node, int TRAIN_NUM, int NUMBER_OF_CLASSES, int FEATURE
	){
	// Naive version
	// threadIdx.x = tree_id, blockIdx.x = rand_feat_i
	int i, y, feat;
	float cut;
	if(!d_is_branch_node[threadIdx.x]){
		return;
	}
	feat = d_random_feats[index(threadIdx.x, blockIdx.x, feat_per_node)];
	cut = d_random_cuts[index(threadIdx.x, blockIdx.x, feat_per_node)];
	for(i=0; i<NUMBER_OF_CLASSES; i++){
		//tree node class
		d_class_counts_a[ixt(threadIdx.x, blockIdx.x, i, feat_per_node, num_trees)] = 0;
		d_class_counts_b[ixt(threadIdx.x, blockIdx.x, i, feat_per_node, num_trees)] = 0;
	}
	for(i=0; i<TRAIN_NUM; i++){
		if(d_batch_pos[index(threadIdx.x, i, TRAIN_NUM)]==tree_pos){
			y = (int) d_y[i];
			if(d_x[index(i, feat, FEATURE)] < cut){
				d_class_counts_a[ixt(threadIdx.x, blockIdx.x, y, feat_per_node, num_trees)]++;
			}else{
				d_class_counts_b[ixt(threadIdx.x, blockIdx.x, y, feat_per_node, num_trees)]++;
			}
		}
	}
}
void populate_class_counts(
		float* d_x, float* d_y, int* d_class_counts_a, int* d_class_counts_b, 
		int* d_random_feats, float* d_random_cuts,
		int* d_batch_pos, int tree_pos,
		int num_trees, int feat_per_node, int* d_is_branch_node, int TRAIN_NUM, int NUMBER_OF_CLASSES, int FEATURE
	){
	// Naive version
	kernel_populate_class_counts<<<feat_per_node, num_trees>>>(
		d_x, d_y, d_class_counts_a, d_class_counts_b, 
		d_random_feats, d_random_cuts,
		d_batch_pos, tree_pos,
		num_trees, feat_per_node,
		d_is_branch_node, TRAIN_NUM, NUMBER_OF_CLASSES, FEATURE
	);
}

/* === Place Best Features/Cuts === */
__global__ void kernel_place_best_feat_cuts(
		int* d_class_counts_a, int* d_class_counts_b, 
		int* d_random_feats, float* d_random_cuts,
		int* d_best_feats, float* d_best_cuts,
		int feat_per_node, int num_trees, int* d_is_branch_node, int NUMBER_OF_CLASSES
	){
	// Naive version => Can move class_counts into shared memory
	// threadIdx.x = tree_id
	int i, k;
    float best_improvement, best_cut, proxy_improvement;
    int best_feat;
    int total_a, total_b;
    float impurity_a, impurity_b;

	if(!d_is_branch_node[threadIdx.x]){
		return;
	}

    best_improvement = -FLT_MAX;
    best_feat = -1;
    best_cut = 0;
	for(i=0; i<feat_per_node; i++){
        total_a = 0;
        total_b = 0;
        impurity_a = 1;
        impurity_b = 1;
        for(k=0; k<NUMBER_OF_CLASSES; k++){
            total_a += d_class_counts_a[ixt(threadIdx.x, i, k, feat_per_node, num_trees)];
            total_b += d_class_counts_b[ixt(threadIdx.x, i, k, feat_per_node, num_trees)];
        }
        for(k=0; k<NUMBER_OF_CLASSES; k++){
            impurity_a -= pow(((float) d_class_counts_a[
            	ixt(threadIdx.x, i, k, feat_per_node, num_trees)]) / total_a, 2);
            impurity_b -= pow(((float) d_class_counts_b[
            	ixt(threadIdx.x, i, k, feat_per_node, num_trees)]) / total_b, 2);
        }
        proxy_improvement = - total_a * impurity_a - total_b * impurity_b;
        if(proxy_improvement > best_improvement){
            best_feat = d_random_feats[index(threadIdx.x, i, feat_per_node)];
            best_cut = d_random_cuts[index(threadIdx.x, i, feat_per_node)];
            best_improvement = proxy_improvement;
        }
	}
	d_best_feats[threadIdx.x] = best_feat;
	d_best_cuts[threadIdx.x] = best_cut;
}
void place_best_feat_cuts(
		int* d_class_counts_a, int* d_class_counts_b, 
		int* d_random_feats, float* d_random_cuts,
		int* d_best_feats, float* d_best_cuts,
		int feat_per_node, int num_trees, int* d_is_branch_node, int NUMBER_OF_CLASSES
	){
	// Naive version
	kernel_place_best_feat_cuts<<<1, num_trees>>>(
		d_class_counts_a, d_class_counts_b, 
		d_random_feats, d_random_cuts,
		d_best_feats, d_best_cuts,
		feat_per_node, num_trees,
		d_is_branch_node, NUMBER_OF_CLASSES
	);
}

/* === Update Trees === */
__global__ void kernel_update_trees(
			float* d_trees, int* d_tree_lengths, int tree_pos,
			int* d_best_feats, float* d_best_cuts, int tree_arr_length, int* d_is_branch_node
		){
	// Naive version
	// threadIdx.x = tree_id
	int left_child_pos, right_child_pos, tree_length;

	if(!d_is_branch_node[threadIdx.x]){
		return;
	}

	tree_length = d_tree_lengths[threadIdx.x];
	left_child_pos = tree_length;
	right_child_pos = tree_length + 1;

	// Update tree nodes
	d_trees[ixt(tree_pos, LEFT_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] = left_child_pos;
	d_trees[ixt(tree_pos, RIGHT_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] = right_child_pos;
	d_trees[ixt(tree_pos, FEAT_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] = d_best_feats[threadIdx.x];
	d_trees[ixt(tree_pos, CUT_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] = d_best_cuts[threadIdx.x];
	d_tree_lengths[threadIdx.x] += 2;

	// Prefill child nodes
	d_trees[ixt(left_child_pos, LEFT_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] = left_child_pos;
	d_trees[ixt(left_child_pos, RIGHT_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] = left_child_pos;
	d_trees[ixt(left_child_pos, DEPTH_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] = \
		d_trees[ixt(tree_pos, DEPTH_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] + 1;
	d_trees[ixt(left_child_pos, FEAT_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] = -1;
	d_trees[ixt(left_child_pos, CUT_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] = -1;
	d_trees[ixt(left_child_pos, PRED_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] = -1;

	d_trees[ixt(right_child_pos, LEFT_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] = right_child_pos;
	d_trees[ixt(right_child_pos, RIGHT_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] = right_child_pos;
	d_trees[ixt(right_child_pos, DEPTH_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] = \
		d_trees[ixt(tree_pos, DEPTH_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] + 1;
	d_trees[ixt(right_child_pos, FEAT_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] = -1;
	d_trees[ixt(right_child_pos, CUT_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] = -1;
	d_trees[ixt(right_child_pos, PRED_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)] = -1;
}
void update_trees(
			float* d_trees, int* d_tree_lengths, int tree_pos,
			int* d_best_feats, float* d_best_cuts, int tree_arr_length,
			int num_trees, int* d_is_branch_node
		){
	kernel_update_trees<<<1, num_trees>>>(
		d_trees, d_tree_lengths, tree_pos,
		d_best_feats, d_best_cuts, tree_arr_length, d_is_branch_node
	);
}

/* === Evaluate === */
__global__ void kernel_raw_predict(
			float *d_raw_pred_y, float* d_trees, int* d_batch_pos, int tree_arr_length, int x_length
		){
	// threadIdx.x = tree_id, blockIdx.x = x_i
	int pos;
	pos = d_batch_pos[index(threadIdx.x, blockIdx.x, x_length)];
	d_raw_pred_y[index(threadIdx.x, blockIdx.x, x_length)] = d_trees[
		ixt(pos, PRED_KEY, threadIdx.x, NUM_FIELDS, tree_arr_length)];
}
void raw_predict(
			float *d_raw_pred_y, float* d_trees, int* d_batch_pos, int tree_arr_length, int x_length,
			int num_trees
		){
	kernel_raw_predict<<<x_length, num_trees>>>(
		d_raw_pred_y, d_trees, d_batch_pos, tree_arr_length, x_length
	);
}
void predict(float* pred_y, float* raw_pred_y, int x_length, int num_trees, int NUMBER_OF_CLASSES){
	int *class_count_buffer;
	int i, j, k, pred, maximum, maximum_class;
	class_count_buffer = (int *)malloc(NUMBER_OF_CLASSES * sizeof(int));
	for(k=0; k<NUMBER_OF_CLASSES; k++){
		class_count_buffer[k] = 0;
	}
	for(i=0; i<x_length; i++){
		for(j=0; j<num_trees; j++){
			pred = (int) raw_pred_y[index(j, i, x_length)];
			class_count_buffer[pred]++;
		}
		maximum = -1;
		for(k=0; k<NUMBER_OF_CLASSES; k++){
			if(class_count_buffer[k] > maximum){
				maximum = class_count_buffer[k];
				maximum_class = k;
			}
			class_count_buffer[k] = 0;
		}
		pred_y[i] = (float) maximum_class;
	}
}
float evaluate(float* pred_y, float* true_y, int y_length){
	int i;
	float score;
	score = 0;
	for(i=0; i<y_length; i++){
		if((int) pred_y[i] == (int) true_y[i]){
			score += 1;
		}
	}
	score /= y_length;
	return score;
}


int main(int argc, char * argv[]){
	float *dataset_train,*dataset_test;
	float *labels_train,*labels_test;
	int mnist_iris;
	int num_trees;
	int seed;

	if(argc != 4){
		fprintf(stderr, "usage: dataset num_trees seed\n");
		fprintf(stderr, "dataset: MNIST=0, IRIS=1\n");
		fprintf(stderr, "num_trees: <1024\n");
		fprintf(stderr, "seed: int\n");
		exit(1);
	}
	
	mnist_iris = atoi(argv[1]);
	num_trees = atoi(argv[2]);
	seed = atoi(argv[3]);
	srand(seed);

	int TRAIN_NUM,FEATURE,TEST_NUM,NUMBER_OF_CLASSES;
	char file_train_set[50],file_train_label[50],file_test_label[50],file_test_set[50];
	
	if(mnist_iris == 0){
		TRAIN_NUM = 60000;
		TEST_NUM = 10000;
		FEATURE =  784;
		NUMBER_OF_CLASSES = 10;

		dataset_train = (float *)malloc(FEATURE * TRAIN_NUM*sizeof(float));
		labels_train = (float *)malloc(TRAIN_NUM*sizeof(float));
		dataset_test = (float *)malloc(FEATURE * TEST_NUM*sizeof(float));
		labels_test = (float *)malloc(TEST_NUM*sizeof(float));

		strncpy(file_test_set, "data/t10k-images-idx3-ubyte", 50);
		strncpy(file_train_set,"data/train-images-idx3-ubyte",50);
		strncpy(file_train_label, "data/train-labels-idx1-ubyte",50);
		strncpy(file_test_label,"data/t10k-labels-idx1-ubyte",50);

		readData(dataset_train,labels_train,file_train_set,file_train_label);
		readData(dataset_test,labels_test,file_test_set,file_test_label);

	}else if(mnist_iris == 1){
		TRAIN_NUM = 100;
		TEST_NUM = 50;
		FEATURE =  4;
		NUMBER_OF_CLASSES = 3;

		dataset_train = (float *)malloc(FEATURE * TRAIN_NUM*sizeof(float));
		labels_train = (float *)malloc(TRAIN_NUM*sizeof(float));
		dataset_test = (float *)malloc(FEATURE * TEST_NUM*sizeof(float));
		labels_test = (float *)malloc(TEST_NUM*sizeof(float));
		strncpy(file_train_set, "data/iris_train.data",50);
		strncpy(file_test_set,"data/iris_test.data",50);
		read_csv_iris(dataset_train,labels_train,TRAIN_NUM,file_train_set);
		read_csv_iris(dataset_test,labels_test,TEST_NUM,file_test_set);

	}else if(mnist_iris == 2){
		TRAIN_NUM = 160000;
		TEST_NUM = 40000;
		char file_train_set[] = "/home/jhancoc4/medicare-data/2019-samples/cb-encoded/part-b-2013-2019-cb-encoded-train.csv";
		char file_test_set[] = "/home/jhancoc4/medicare-data/2019-samples/cb-encoded/part-b-2013-2019-cb-encoded-test.csv";

		FEATURE =  get_num_features(file_train_set);
		NUMBER_OF_CLASSES = 2;

		dataset_train = (float *)malloc(FEATURE * TRAIN_NUM*sizeof(float));
		labels_train = (float *)malloc(TRAIN_NUM*sizeof(float));
		dataset_test = (float *)malloc(FEATURE * TEST_NUM*sizeof(float));
		labels_test = (float *)malloc(TEST_NUM*sizeof(float));
		read_csv(dataset_train,labels_train,TRAIN_NUM,file_train_set);

		read_csv(dataset_test,labels_test,TEST_NUM,file_test_set);
	}


	float *dataset_train_T;
	dataset_train_T = (float *)malloc(TRAIN_NUM * FEATURE * sizeof(float));
	copy_transpose(dataset_train_T, dataset_train, TRAIN_NUM, FEATURE);

	float *d_trees;
	int *tree_arr_length;
	int *d_tree_lengths;
	int *max_tree_length, *d_max_tree_length;
	int feat_per_node;
	int *d_num_valid_feat;
	int tree_pos;
	int *batch_pos, *d_batch_pos; // NUM_TREES * TRAIN_NUM
	int *d_is_branch_node;
	int *tree_is_done, *d_tree_is_done;
	float *d_min_max_buffer;
	int *d_random_feats_idx;
	int *d_random_feats;
	float *d_random_cuts;
	int *d_class_counts_a, *d_class_counts_b;
	int *d_best_feats;
	float *d_best_cuts;
	float *d_x, *d_y;
	float *d_x_T;
	float *pred_y, *raw_pred_y, *d_raw_pred_y;
	hiprandState_t* curand_states;

	// Assumption: num_trees < maxNumBlocks, maxThreadsPerBlock

	tree_arr_length = (int *)malloc(sizeof(int));
	*tree_arr_length = 8;
	max_tree_length = (int *)malloc(sizeof(int));

	feat_per_node = (int) ceil(sqrt(FEATURE));

	batch_pos = (int *)malloc(num_trees * TRAIN_NUM *sizeof(float));
	tree_is_done = (int *)malloc(num_trees * sizeof(int));
	
	hipDeviceProp_t dev_prop;
	hipGetDeviceProperties(&dev_prop, 0);
	hipMalloc((void **) &d_trees, num_trees * NUM_FIELDS * (*tree_arr_length) *sizeof(float));
	hipMalloc((void **) &d_tree_lengths, num_trees * sizeof(int));
	hipMalloc((void **) &d_max_tree_length, sizeof(int));
	hipMalloc((void **) &d_batch_pos, num_trees * TRAIN_NUM *sizeof(float));
	hipMalloc((void **) &d_is_branch_node, num_trees * sizeof(int));
	hipMalloc((void **) &d_tree_is_done, num_trees * sizeof(int));
	hipMalloc((void **) &d_min_max_buffer, num_trees * FEATURE * 2 *sizeof(float));
	hipMalloc((void **) &d_num_valid_feat, num_trees *sizeof(int));
	hipMalloc((void **) &d_random_feats_idx, num_trees * FEATURE * sizeof(int));
	hipMalloc((void **) &d_random_feats, num_trees * feat_per_node * sizeof(int));
	hipMalloc((void **) &d_random_cuts, num_trees * feat_per_node * sizeof(float));
	hipMalloc((void **) &d_best_feats, num_trees * sizeof(int));
	hipMalloc((void **) &d_best_cuts, num_trees * sizeof(float));
	hipMalloc((void **) &d_class_counts_a, num_trees * feat_per_node * NUMBER_OF_CLASSES *sizeof(int));
	hipMalloc((void **) &d_class_counts_b, num_trees * feat_per_node * NUMBER_OF_CLASSES *sizeof(int));
	hipMalloc((void **) &d_x, TRAIN_NUM * FEATURE *sizeof(float));
	hipMalloc((void **) &d_y, TRAIN_NUM *sizeof(float));
	hipMalloc((void **) &d_x_T, TRAIN_NUM * FEATURE *sizeof(float));
	hipMemcpy(d_x, dataset_train, TRAIN_NUM * FEATURE *sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, labels_train, TRAIN_NUM *sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_x_T, dataset_train_T, TRAIN_NUM * FEATURE *sizeof(float), hipMemcpyHostToDevice);


	hipMalloc((void**) &curand_states, num_trees * sizeof(hiprandState));
	init_random<<<1, num_trees>>>(1337, curand_states);

	initialize_trees(d_trees, num_trees, *tree_arr_length, d_tree_lengths);
	initialize_batch_pos(d_batch_pos, TRAIN_NUM, num_trees, dev_prop);

	for(tree_pos=0; tree_pos<100000; tree_pos++){
		printf("* ================== TREE POS -[ %d ]- ================== *\n", tree_pos);

		refresh_tree_is_done(d_tree_lengths, d_tree_is_done, tree_pos, num_trees);
		if(check_forest_done(d_tree_is_done, tree_is_done, num_trees)){
			printf("DONE\n");
			break;
		}

		maybe_expand(
			&d_trees, num_trees, tree_arr_length, d_tree_lengths, max_tree_length, d_max_tree_length);

		batch_advance_trees(d_trees, d_x, TRAIN_NUM, *tree_arr_length, num_trees, d_batch_pos, dev_prop, TRAIN_NUM, FEATURE);

		check_node_termination(
			d_trees, *tree_arr_length, 
			d_y, d_batch_pos, tree_pos,
			d_is_branch_node, d_tree_is_done,
			num_trees, TRAIN_NUM
		);
		collect_min_max(
			d_x_T, d_batch_pos, tree_pos, num_trees, TRAIN_NUM,
			d_min_max_buffer, d_is_branch_node, dev_prop, TRAIN_NUM, FEATURE
		);
		collect_num_valid_feat(
			d_num_valid_feat, d_random_feats_idx,
			d_min_max_buffer, num_trees, d_is_branch_node, dev_prop, FEATURE
		);
		populate_feat_cut(
			d_random_feats, d_random_cuts, 
			d_random_feats_idx, d_num_valid_feat, 
			d_min_max_buffer, feat_per_node, num_trees, 
			d_is_branch_node, curand_states, FEATURE
		);
		populate_class_counts(
			d_x, d_y, d_class_counts_a, d_class_counts_b, 
			d_random_feats, d_random_cuts,
			d_batch_pos, tree_pos,
			num_trees, feat_per_node, 
			d_is_branch_node, TRAIN_NUM, NUMBER_OF_CLASSES, FEATURE
		);
		place_best_feat_cuts(
			d_class_counts_a, d_class_counts_b, 
			d_random_feats, d_random_cuts,
			d_best_feats, d_best_cuts,
			feat_per_node, num_trees, 
			d_is_branch_node, NUMBER_OF_CLASSES
		);
		update_trees(
			d_trees, d_tree_lengths, tree_pos,
			d_best_feats, d_best_cuts, *tree_arr_length,
			num_trees, 
			d_is_branch_node
		);
		hipDeviceSynchronize();
	}

	printf("================= DONE TRAINING =================\n");
	/* === TEST === */
	hipFree(d_batch_pos);
	free(batch_pos);
	hipMalloc((void **) &d_batch_pos, num_trees * TEST_NUM * sizeof(float));
	pred_y = (float *)malloc(TEST_NUM * sizeof(float));
	raw_pred_y = (float *)malloc(num_trees * TEST_NUM * sizeof(float));

	hipFree(d_x);
	hipMalloc((void **) &d_x, TEST_NUM * FEATURE * sizeof(float));
	hipMalloc((void **) &d_raw_pred_y, num_trees * TEST_NUM * sizeof(float));
	hipMemcpy(d_x, dataset_test, TEST_NUM * FEATURE * sizeof(float), hipMemcpyHostToDevice);

	initialize_batch_pos(
		d_batch_pos, TEST_NUM, num_trees, dev_prop
	);
	batch_traverse_trees(
		d_trees, d_x, TEST_NUM, num_trees, *tree_arr_length, d_batch_pos, dev_prop, FEATURE
	);
	hipMemcpy(d_x, dataset_test, TEST_NUM * FEATURE * sizeof(float), hipMemcpyHostToDevice);
	raw_predict(d_raw_pred_y, d_trees, d_batch_pos, *tree_arr_length, TEST_NUM, num_trees);
	hipMemcpy(raw_pred_y, d_raw_pred_y, num_trees * TEST_NUM * sizeof(float), hipMemcpyDeviceToHost);
	predict(pred_y, raw_pred_y, TEST_NUM, num_trees, NUMBER_OF_CLASSES);

	printf("Test Accuracy: %f\n", evaluate(pred_y, labels_test, TEST_NUM));
	debug();
}
